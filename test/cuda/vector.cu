#include "hip/hip_runtime.h"

#include "core/adt/vector.hpp"
#include "test/test.hpp"

struct vec3
{
    float x, y, z;
    float *c;
    vec3() : x(0), y(0), z(0), c(nullptr) {}
    vec3(const vec3 &v) : x(v.x), y(v.y), z(v.z), c(v.c + 1) {}
    vec3(float x, float y = 1.f, float z = 2.f) : x(x), y(y), z(z), c(nullptr)
    {
    }
};

CUDAKERNEL test1(int *a)
{
    mj::vector<int, 20> v(10, 2);
    for (auto i : v)
        atomicAdd(a, i);
    v[1] = 0;
    for (auto i : v)
        atomicAdd(a + 1, i);
    v.push_back(3);
    for (auto i : v)
        atomicAdd(a + 2, i);
    atomicAdd(a + 3, v.size());

    v.erase(v.begin() + 1); // erase the 0
    for (auto i : v)
        atomicAdd(a + 4, i);

    v.erase(v.begin(), v.begin() + 2); // erase two 2s
    for (auto i : v)
        atomicAdd(a + 5, i);
    v.insert(v.begin() + 4, 21);
    atomicAdd(a + 6, v.size());
    atomicAdd(a + 7, v[4]);

    v.pop_back();
    for (auto i : v)
        atomicAdd(a + 8, i);
    v.clear();
    atomicAdd(a + 9, v.size());
}

CUDAKERNEL test2(vec3 *a)
{
    mj::vector<vec3, 20> v; // default constructor
}

int main()
{
    int *da1, a1[10];
    hipMalloc(&da1, sizeof(int) * 10);
    hipMemset(da1, 0, sizeof(int) * 10);
    test1<<<10, 10>>>(da1);
    hipMemcpy(a1, da1, sizeof(int) * 10, hipMemcpyDeviceToHost);
    assert(a1[0] == 2000, "fill initialization");
    assert(a1[1] == 1800, "operator[] nonconst");
    assert(a1[2] == 2100, "push_back");
    assert(a1[3] == 1100, "size");
    assert(a1[4] == 2100, "erase");
    assert(a1[5] == 1700, "erase range");
    assert(a1[6] == 900, "size after erase and inserts");
    assert(a1[7] == 2100, "insert");
    assert(a1[8] == 4100, "pop_back");
    assert(a1[9] == 0, "clear");
    hipFree(da1);

    vec3 *da2, a2[10];
    hipMalloc(&da2, sizeof(int) * 10);
    hipMemset(da2, 0, sizeof(int) * 10);
    test2<<<10, 10>>>(da2);
    hipMemcpy(a2, da2, sizeof(int) * 10, hipMemcpyDeviceToHost);
    hipFree(da2);
}